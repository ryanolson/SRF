#include "hip/hip_runtime.h"

#include "radar.hpp"

#include "mrc/cuda/common.hpp"
#include "mrc/cuda/sync.hpp"

CudaStream::CudaStream()
{
    MRC_CHECK_CUDA(hipStreamCreate(&stream));
    LOG(INFO) << "stream initialized";
}

CudaStream::~CudaStream()
{
    MRC_CHECK_CUDA(hipStreamSynchronize(stream));
    MRC_CHECK_CUDA(hipStreamDestroy(stream));
}

StreamProvider::StreamProvider(mrc::data::Reusable<CudaStream> stream_handle) :
  m_stream_handle(std::move(stream_handle))
{}

mrc::data::Reusable<CudaStream> StreamProvider::release_stream()
{
    return std::move(m_stream_handle);
}

ThreePulseCancellerData::ThreePulseCancellerData(mrc::data::Reusable<tensor_t<ComplexType, 3>> _inputView,
                                                 mrc::data::Reusable<CudaStream> _stream) :
  inputView(std::move(_inputView)),
  StreamProvider(std::move(_stream))
{}

DopplerData::DopplerData(mrc::data::Reusable<tensor_t<ComplexType, 3>> _tpcView,
                         tensor_t<ftype, 1>* _cancelMask,
                         mrc::data::Reusable<CudaStream> _stream) :
  tpcView(std::move(_tpcView)),
  cancelMask(_cancelMask),
  StreamProvider(std::move(_stream))
{}

CFARData::CFARData(mrc::data::Reusable<tensor_t<ComplexType, 3>> _tpcView, mrc::data::Reusable<CudaStream> _stream) :
  tpcView(std::move(_tpcView)),
  StreamProvider(std::move(_stream))
{}

PulseCompressionOp::PulseCompressionOp(RadarConfig radar_config, const RuntimeOptions& opts) :
  radar_config(std::move(radar_config)),
  m_opts(opts)
{
    LOG(INFO) << "pulse compress op constructor start";

    LOG(INFO) << "numCudaStreams: " << m_opts.num_streams;
    m_stream_pool = mrc::data::ReusablePool<CudaStream>::create(16);
    for (int i = 0; i < m_opts.num_streams; i++)
    {
        m_stream_pool->emplace();
    }
    auto stream_handle = m_stream_pool->await_item();

    norms = new tensor_t<ftype, 0>();

    LOG(INFO) << "numPulses: " << radar_config.numPulses;
    LOG(INFO) << "numSamples: " << radar_config.numSamples;
    LOG(INFO) << "numChannels: " << radar_config.numChannels;
    LOG(INFO) << "waveformLength: " << radar_config.waveformLength;

    numSamplesRnd = 1;
    while (numSamplesRnd < radar_config.numSamples)
    {
        numSamplesRnd *= 2;
    }
    LOG(INFO) << "numSamplesRnd: " << numSamplesRnd;

    waveformView = new tensor_t<ComplexType, 1>({numSamplesRnd});

    m_input_pool = mrc::data::ReusablePool<tensor_t<ComplexType, 3>>::create(16);
    for (int i = 0; i < m_opts.num_streams; i++)
    {
        auto view = std::unique_ptr<tensor_t<ComplexType, 3>>(
            new tensor_t<ComplexType, 3>({radar_config.numChannels, radar_config.numPulses, numSamplesRnd}));
        hipMemset(view->Data(), 0, view->TotalSize() * sizeof(ComplexType));
        view->PrefetchDevice(stream_handle->stream);
        m_input_pool->add_item(std::move(view));
    }
    // inputView = new tensor_t<ComplexType, 3>({radar_config.numChannels, radar_config.numPulses, numSamplesRnd});

    hipMemset(waveformView->Data(), 0, numSamplesRnd * sizeof(ComplexType));
    // hipMemset(inputView->Data(), 0, inputView->TotalSize() * sizeof(ComplexType));

    waveformView->PrefetchDevice(stream_handle->stream);
    // inputView->PrefetchDevice(stream_handle->stream);

    MRC_CHECK_CUDA(hipStreamSynchronize(stream_handle->stream));
    LOG(INFO) << "pulse compress op constructor finish";
    stream_handle.release();
}

void PulseCompressionOp::data_source(rxcpp::subscriber<source_type_t>& sub)
{
    auto& context = mrc::runnable::Context::get_runtime_context();

    m_opts.m_start = std::chrono::system_clock::now();

    while (sub.is_subscribed() && m_current_count < m_opts.iterations)
    {
        auto stream_handle = m_stream_pool->await_item();
        auto inputView     = m_input_pool->await_item();

        // Reshape waveform to be waveformLength
        auto waveformPart = waveformView->Slice({0}, {radar_config.waveformLength});
        auto waveformT =
            waveformView->template Clone<3>({radar_config.numChannels, radar_config.numPulses, matxKeepDim});
        auto waveformFull = waveformView->Slice({0}, {numSamplesRnd});

        auto x = *inputView;

        // create waveform (assuming waveform is the same for every pulse)
        // this allows us to precompute waveform in frequency domain
        // Apply a Hamming window to the waveform to suppress sidelobes. Other
        // windows could be used as well (e.g., Taylor windows). Ultimately, it is
        // just an element-wise weighting by a pre-computed window function.
        (waveformPart = waveformPart * hamming<0>({radar_config.waveformLength})).run(stream_handle->stream);

        // compute L2 norm
        sum(*norms, norm(waveformPart), stream_handle->stream);
        (*norms = sqrt(*norms)).run(stream_handle->stream);

        (waveformPart = waveformPart / *norms).run(stream_handle->stream);
        fft(waveformFull, waveformPart, 0, stream_handle->stream);
        (waveformFull = conj(waveformFull)).run(stream_handle->stream);

        fft(x, x, 0, stream_handle->stream);
        (x = x * waveformT).run(stream_handle->stream);
        ifft(x, x, 0, stream_handle->stream);

        sub.on_next(std::make_unique<ThreePulseCancellerData>(std::move(inputView), std::move(stream_handle)));
        context.yield();
        m_current_count++;
    }
}

ThreePulseCancellerOp::ThreePulseCancellerOp(RadarConfig radar_config, const RuntimeOptions& opts) :
  radar_config(std::move(radar_config)),
  m_opts(opts)
{
    numPulsesRnd = 1;
    while (numPulsesRnd <= radar_config.numPulses)
    {
        numPulsesRnd *= 2;
    }

    numCompressedSamples = radar_config.numSamples - radar_config.waveformLength + 1;

    m_tpc_pool = mrc::data::ReusablePool<tensor_t<ComplexType, 3>>::create(16);
    for (int i = 0; i < m_opts.num_streams; i++)
    {
        auto view = std::unique_ptr<tensor_t<ComplexType, 3>>(
            new tensor_t<ComplexType, 3>({radar_config.numChannels, numPulsesRnd, numCompressedSamples}));
        hipMemset(view->Data(), 0, view->TotalSize() * sizeof(ComplexType));
        view->PrefetchDevice(0);
        m_tpc_pool->add_item(std::move(view));
    }

    // tpcView    = new tensor_t<ComplexType, 3>({radar_config.numChannels, numPulsesRnd, numCompressedSamples});
    cancelMask = new tensor_t<ftype, 1>({3});
    cancelMask->SetVals({1, -2, 1});

    // hipMemset(tpcView->Data(), 0, tpcView->TotalSize() * sizeof(ComplexType));

    // tpcView->PrefetchDevice(0);
    cancelMask->PrefetchDevice(0);

    MRC_CHECK_CUDA(hipDeviceSynchronize());
}

void ThreePulseCancellerOp::on_data(sink_type_t&& tpc_data, rxcpp::subscriber<source_type_t>& subscriber)
{
    auto tpcView = m_tpc_pool->await_item();

    auto x = tpc_data->inputView->Permute({0, 2, 1}).Slice(
        {0, 0, 0}, {radar_config.numChannels, numCompressedSamples, radar_config.numPulses});
    auto xo = tpcView->Permute({0, 2, 1}).Slice(
        {0, 0, 0}, {radar_config.numChannels, numCompressedSamples, radar_config.numPulses});
    conv1d(xo, x, *cancelMask, matxConvCorrMode_t::MATX_C_MODE_SAME, tpc_data->stream());

    subscriber.on_next(std::make_unique<DopplerData>(std::move(tpcView), cancelMask, tpc_data->release_stream()));
    tpc_data->inputView.release();
    mrc::runnable::Context::get_runtime_context().yield();
}
DopplerOp::DopplerOp(RadarConfig radar_config) : radar_config(std::move(radar_config))
{
    numCompressedSamples = radar_config.numSamples - radar_config.waveformLength + 1;
}
void DopplerOp::on_data(sink_type_t&& data, rxcpp::subscriber<source_type_t>& subscriber)
{
    auto dop_data = std::move(data);

    const index_t cpulses = radar_config.numPulses - (dop_data->cancelMask->Size(0) - 1);

    auto xc = dop_data->tpcView->Slice({0, 0, 0}, {radar_config.numChannels, cpulses, numCompressedSamples});
    auto xf = dop_data->tpcView->Permute({0, 2, 1});

    (xc = xc * hamming<1>({radar_config.numChannels,
                           radar_config.numPulses - (dop_data->cancelMask->Size(0) - 1),
                           numCompressedSamples}))
        .run(dop_data->stream());
    fft(xf, xf, 0, dop_data->stream());

    subscriber.on_next(std::make_unique<CFARData>(std::move(dop_data->tpcView), dop_data->release_stream()));
    mrc::runnable::Context::get_runtime_context().yield();
}
CFAROp::CFAROp(RadarConfig radar_config, const RuntimeOptions& opts) :
  radar_config(std::move(radar_config)),
  m_opts(opts)
{
    numPulsesRnd = 1;
    while (numPulsesRnd <= radar_config.numPulses)
    {
        numPulsesRnd *= 2;
    }

    numCompressedSamples = radar_config.numSamples - radar_config.waveformLength + 1;

    normT = new tensor_t<ftype, 3>(
        {radar_config.numChannels, numPulsesRnd + cfarMaskY - 1, numCompressedSamples + cfarMaskX - 1});
    ba = new tensor_t<ftype, 3>(
        {radar_config.numChannels, numPulsesRnd + cfarMaskY - 1, numCompressedSamples + cfarMaskX - 1});
    dets         = new tensor_t<int, 3>({radar_config.numChannels, numPulsesRnd, numCompressedSamples});
    xPow         = new tensor_t<ftype, 3>({radar_config.numChannels, numPulsesRnd, numCompressedSamples});
    cfarMaskView = new tensor_t<ftype, 2>({cfarMaskY, cfarMaskX});

    // Mask for cfar detection
    // G == guard, R == reference, C == CUT
    // mask = [
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R G G G R ;
    //    R G C G R ;
    //    R G G G R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ;
    //    R R R R R ];
    //  }
    cfarMaskView->SetVals({{1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
                           {1, 1, 1, 1, 1, 0, 0, 0, 1, 1, 1, 1, 1},
                           {1, 1, 1, 1, 1, 0, 0, 0, 1, 1, 1, 1, 1},
                           {1, 1, 1, 1, 1, 0, 0, 0, 1, 1, 1, 1, 1},
                           {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1}});

    // Pre-process CFAR convolution
    conv2d(*normT,
           ones({radar_config.numChannels, numPulsesRnd, numCompressedSamples}),
           *cfarMaskView,
           matxConvCorrMode_t::MATX_C_MODE_FULL,
           0);

    ba->PrefetchDevice(0);
    normT->PrefetchDevice(0);
    cfarMaskView->PrefetchDevice(0);
    dets->PrefetchDevice(0);
    xPow->PrefetchDevice(0);

    MRC_CHECK_CUDA(hipDeviceSynchronize());
}
void CFAROp::on_data(sink_type_t&& cfar_data)
{
    (*xPow = norm(*cfar_data->tpcView)).run(cfar_data->stream());

    // Estimate the background average power in each cell
    // background_averages = conv2(Xpow, mask, 'same') ./ norm;
    conv2d(*ba, *xPow, *cfarMaskView, matxConvCorrMode_t::MATX_C_MODE_FULL, cfar_data->stream());

    // Computing number of cells contributing to each cell.
    // This can be done with a convolution of the cfarMask with
    // ones.
    // norm = conv2(ones(size(X)), mask, 'same');
    auto normTrim =
        normT->Slice({0, cfarMaskY / 2, cfarMaskX / 2},
                     {radar_config.numChannels, numPulsesRnd + cfarMaskY / 2, numCompressedSamples + cfarMaskX / 2});

    auto baTrim =
        ba->Slice({0, cfarMaskY / 2, cfarMaskX / 2},
                  {radar_config.numChannels, numPulsesRnd + cfarMaskY / 2, numCompressedSamples + cfarMaskX / 2});
    (baTrim = baTrim / normTrim).run(cfar_data->stream());

    // The scalar alpha is used as a multiplier on the background averages
    // to achieve a constant false alarm rate (under certain assumptions);
    // it is based upon the desired probability of false alarm (Pfa) and
    // number of reference cells used to estimate the background for the
    // CUT. For the purposes of computation, it can be assumed as a given
    // constant, although it does vary at the edges due to the different
    // training windows.
    // Declare a detection if the power exceeds the background estimate
    // times alpha for a particular cell.
    // dets(find(Xpow > alpha.*background_averages)) = 1;

    // These 2 branches are functionally equivalent.  A custom op is more
    // efficient as it can avoid repeated loads.
    calcDets(*dets, *xPow, baTrim, normTrim, pfa).run(cfar_data->stream());
    mrc::enqueue_stream_sync_event(cfar_data->stream()).get();
    cfar_data.reset();
    mrc::runnable::Context::get_runtime_context().yield();
}

void CFAROp::on_completed()
{
    m_opts.m_finish = std::chrono::system_clock::now();
}
